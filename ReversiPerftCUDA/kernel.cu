#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "MacrosHell.h"
#include "Position.h"
#include <numeric>
#include <omp.h>

__constant__ uint8_t  d_OUTFLANK[8][ 64];
__constant__ uint8_t  d_FLIPS   [8][256];
__constant__ uint64_t d_STRETCH    [256];
__constant__ uint64_t d_MASK_D     [ 64];
__constant__ uint64_t d_MASK_C     [ 64];

void Initialize()
{
	uint8_t OUTFLANK[8][64];
	uint8_t FLIPS[8][256];
	uint64_t STRETCH[256];
	uint64_t MASK_D[64];
	uint64_t MASK_C[64];

	uint8_t O, k, outf;
	for (uint8_t j = 0; j < 8; j++)
	{
		for (uint8_t i = 0; i < 64; i++)
		{
			outf = 0;
			O = i << 1;

			k = j + 1;
			while (O & (1 << k))
				k++;
			if (k != j + 1) // There was an outflank
				outf |= 1 << k;

			k = j - 1;
			while (O & (1 << k))
				k--;
			if (k != j - 1) // There was an outflank
				outf |= 1 << k;

			OUTFLANK[j][i] = outf;
		}
	}

	for (unsigned int i = 0; i < 8; i++)
		for (unsigned int j = 0; j < 256; j++)
			FLIPS[i][j] = 0;

	// 0000 000X
	FLIPS[0][0x04] = 0x02;
	FLIPS[0][0x08] = 0x06;
	FLIPS[0][0x10] = 0x0E;
	FLIPS[0][0x20] = 0x1E;
	FLIPS[0][0x40] = 0x3E;
	FLIPS[0][0x80] = 0x7E;

	// 0000 00X0
	FLIPS[1][0x08] = 0x04;
	FLIPS[1][0x10] = 0x0C;
	FLIPS[1][0x20] = 0x1C;
	FLIPS[1][0x40] = 0x3C;
	FLIPS[1][0x80] = 0x7C;

	// 0000 0X00
	FLIPS[2][0x01] = 0x02;
	FLIPS[2][0x10] = 0x08;
	FLIPS[2][0x11] = 0x0A;
	FLIPS[2][0x20] = 0x18;
	FLIPS[2][0x21] = 0x1A;
	FLIPS[2][0x40] = 0x38;
	FLIPS[2][0x41] = 0x3A;
	FLIPS[2][0x80] = 0x78;
	FLIPS[2][0x81] = 0x7A;

	// 0000 X000
	FLIPS[3][0x01] = 0x06;
	FLIPS[3][0x02] = 0x04;
	FLIPS[3][0x20] = 0x10;
	FLIPS[3][0x21] = 0x16;
	FLIPS[3][0x22] = 0x14;
	FLIPS[3][0x40] = 0x30;
	FLIPS[3][0x41] = 0x36;
	FLIPS[3][0x42] = 0x34;
	FLIPS[3][0x80] = 0x70;
	FLIPS[3][0x81] = 0x76;
	FLIPS[3][0x82] = 0x74;

	// 000X 0000
	FLIPS[4][0x01] = 0x0E;
	FLIPS[4][0x02] = 0x0C;
	FLIPS[4][0x04] = 0x08;
	FLIPS[4][0x40] = 0x20;
	FLIPS[4][0x41] = 0x2E;
	FLIPS[4][0x42] = 0x2C;
	FLIPS[4][0x44] = 0x28;
	FLIPS[4][0x80] = 0x60;
	FLIPS[4][0x81] = 0x6E;
	FLIPS[4][0x82] = 0x6C;
	FLIPS[4][0x84] = 0x68;

	// 00X0 0000
	FLIPS[5][0x01] = 0x1E;
	FLIPS[5][0x02] = 0x1C;
	FLIPS[5][0x04] = 0x18;
	FLIPS[5][0x08] = 0x10;
	FLIPS[5][0x80] = 0x40;
	FLIPS[5][0x81] = 0x5E;
	FLIPS[5][0x82] = 0x5C;
	FLIPS[5][0x84] = 0x58;
	FLIPS[5][0x88] = 0x50;

	// 0X00 0000
	FLIPS[6][0x01] = 0x3E;
	FLIPS[6][0x02] = 0x3C;
	FLIPS[6][0x04] = 0x38;
	FLIPS[6][0x08] = 0x30;
	FLIPS[6][0x10] = 0x20;

	// X000 0000
	FLIPS[7][0x01] = 0x7E;
	FLIPS[7][0x02] = 0x7C;
	FLIPS[7][0x04] = 0x78;
	FLIPS[7][0x08] = 0x70;
	FLIPS[7][0x10] = 0x60;
	FLIPS[7][0x20] = 0x40;

	for (uint64_t i = 0; i < 256; i++)
		STRETCH[i] = ((i * 0x0102040810204080ULL) & 0x0101010101010101ULL) * 0xFFULL;

	for (unsigned int i = 0; i < 64; i++)
	{
		unsigned int L = i % 8;
		unsigned int N = i / 8;
		if (L > N) MASK_D[i] = 0x8040201008040201ULL >> ((L - N) * 8);
		else MASK_D[i] = 0x8040201008040201ULL << ((N - L) * 8);
	}

	for (unsigned int i = 0; i < 64; i++)
	{
		unsigned int L = i % 8;
		unsigned int N = i / 8;
		if (N + L > 7) MASK_C[i] = 0x0102040810204080ULL << ((N + L - 7) * 8);
		else MASK_C[i] = 0x0102040810204080ULL >> (-(N + L - 7) * 8);
	}

	for (int i = 0; i < 2; i++)
	{
		hipSetDevice(i);
		hipMemcpyToSymbol(HIP_SYMBOL(d_OUTFLANK), OUTFLANK, sizeof(uint8_t) * 8 * 64);
		hipMemcpyToSymbol(HIP_SYMBOL(d_FLIPS), FLIPS, sizeof(uint8_t) * 8 * 256);
		hipMemcpyToSymbol(HIP_SYMBOL(d_STRETCH), STRETCH, sizeof(uint64_t) * 256);
		hipMemcpyToSymbol(HIP_SYMBOL(d_MASK_D), MASK_D, sizeof(uint64_t) * 64);
		hipMemcpyToSymbol(HIP_SYMBOL(d_MASK_C), MASK_C, sizeof(uint64_t) * 64);
	}
}

template <const unsigned int dir>
__device__ __inline__ uint64_t CUDA_get_some_moves(const uint64_t P, const uint64_t mask)
{
	// kogge-stone parallel prefix
	// 12 x SHIFT, 10 x AND, 7 x OR
	// = 29 OPs
	uint64_t flip_l, flip_r;
	uint64_t mask_l, mask_r;

	flip_l = mask & (P << dir);
	flip_r = mask & (P >> dir);

	flip_l |= mask & (flip_l << dir);
	flip_r |= mask & (flip_r >> dir);

	mask_l = mask & (mask << dir);
	mask_r = mask & (mask >> dir);

	flip_l |= mask_l & (flip_l << (dir * 2));
	flip_r |= mask_r & (flip_r >> (dir * 2));

	flip_l |= mask_l & (flip_l << (dir * 2));
	flip_r |= mask_r & (flip_r >> (dir * 2));

	return (flip_l << dir) | (flip_r >> dir);
}

__device__ uint64_t CUDA_HasMoves(const CPosition& pos)
{
	const uint64_t empties = pos.Empties();
	if (CUDA_get_some_moves<1>(pos.GetP(), pos.GetO() & 0x7E7E7E7E7E7E7E7EULL) & empties) return 1;
	if (CUDA_get_some_moves<8>(pos.GetP(), pos.GetO() & 0x00FFFFFFFFFFFF00ULL) & empties) return 1;
	if (CUDA_get_some_moves<7>(pos.GetP(), pos.GetO() & 0x007E7E7E7E7E7E00ULL) & empties) return 1;
	if (CUDA_get_some_moves<9>(pos.GetP(), pos.GetO() & 0x007E7E7E7E7E7E00ULL) & empties) return 1;
	return 0;
}

__device__ __inline__ uint64_t CUDA_flip_h(const CPosition& pos, const uint8_t move)
{
	const uint64_t O = (pos.GetO() >> ((move & 0xF8) + 1)) & 0x3FULL;
	const uint64_t P = (pos.GetP() >> (move & 0xF8)) & 0xFFULL;
	const uint64_t outflank = d_OUTFLANK[move & 7][O] & P;
	return static_cast<uint64_t>(d_FLIPS[move & 7][outflank]) << (move & 0xF8);
}

__device__ __inline__ uint64_t CUDA_flip_v(const CPosition& pos, const uint8_t move)
{
	const uint64_t O = ((pos.GetO() & (0x0001010101010100ULL << (move & 7))) * (0x0102040810204080ULL >> (move & 7))) >> 57;
	const uint64_t P = ((pos.GetP() & (0x0101010101010101ULL << (move & 7))) * (0x0102040810204080ULL >> (move & 7))) >> 56;
	const uint64_t outflank = d_OUTFLANK[(move >> 3)][O] & P;
	return d_STRETCH[d_FLIPS[(move >> 3)][outflank]] & (0x0101010101010101ULL << (move & 7));
}

__device__ __inline__ uint64_t CUDA_flip_d(const CPosition& pos, const uint8_t move)
{
	const uint64_t O = ((pos.GetO() & d_MASK_D[move] & 0x007E7E7E7E7E7E00ULL) * 0x0101010101010101ULL) >> 57;
	const uint64_t P = ((pos.GetP() & d_MASK_D[move]) * 0x0101010101010101ULL) >> 56;
	const uint64_t outflank = d_OUTFLANK[move & 7][O] & P;
	return (d_FLIPS[move & 7][outflank] * 0x0101010101010101ULL) & d_MASK_D[move];
}

__device__ __inline__ uint64_t CUDA_flip_c(const CPosition& pos, const uint8_t move)
{
	const uint64_t O = ((pos.GetO() & d_MASK_C[move] & 0x007E7E7E7E7E7E00ULL) * 0x0101010101010101ULL) >> 57;
	const uint64_t P = ((pos.GetP() & d_MASK_C[move]) * 0x0101010101010101ULL) >> 56;
	const uint64_t outflank = d_OUTFLANK[move & 7][O] & P;
	return (d_FLIPS[move & 7][outflank] * 0x0101010101010101ULL) & d_MASK_C[move];
}

__device__ uint64_t CUDA_flip(const CPosition& pos, const uint8_t move)
{
	const auto h = CUDA_flip_h(pos, move);
	const auto v = CUDA_flip_v(pos, move);
	const auto d = CUDA_flip_d(pos, move);
	const auto c = CUDA_flip_c(pos, move);
	return h | v | d | c;
}

__device__ uint32_t GPUperft2(const CPosition& pos)
{
	auto moves = PossibleMoves(pos);
	if (moves.empty())
		return PossibleMoves(pos.PlayPass()).size();

	uint32_t sum = 0;
	while (!moves.empty())
	{
		auto move = moves.ExtractMove();
		uint64_t flipped = CUDA_flip(pos, move);
		const auto next_pos = pos.Play(move, flipped);
		auto next_moves = PossibleMoves(next_pos);
		if (next_moves.empty())
			sum += CUDA_HasMoves(next_pos.PlayPass());
		else
			sum += next_moves.size();
	}
	return sum;
}

__device__ uint32_t GPUperft3(const CPosition& pos)
{
	auto moves = PossibleMoves(pos);
	if (moves.empty())
	{
		auto pos_pass = pos.PlayPass();
		if (CUDA_HasMoves(pos_pass))
			return GPUperft2(pos_pass);
		return 0;
	}

	uint32_t sum = 0;
	while (!moves.empty())
	{
		auto move = moves.ExtractMove();
		uint64_t flipped = CUDA_flip(pos, move);
		sum += GPUperft2(pos.Play(move, flipped));
	}
	return sum;
}

__device__ uint32_t GPUperft4(const CPosition& pos)
{
	auto moves = PossibleMoves(pos);
	if (moves.empty())
	{
		auto pos_pass = pos.PlayPass();
		if (CUDA_HasMoves(pos_pass))
			return GPUperft3(pos_pass);
		return 0;
	}

	uint32_t sum = 0;
	while (!moves.empty())
	{
		auto move = moves.ExtractMove();
		uint64_t flipped = CUDA_flip(pos, move);
		sum += GPUperft3(pos.Play(move, flipped));
	}
	return sum;
}

__global__ void kernel(const CPosition * pos, uint32_t * result, uint64_t size)
{
	//volatile __shared__ uint32_t sdata[blockSize];
	const std::size_t tid = threadIdx.x;
	const std::size_t gridSize = blockDim.x * gridDim.x;

	for (int i = tid + blockIdx.x * blockDim.x; i < size; i += gridSize)
	{
		result[i] = GPUperft4(pos[i]);
	}
}

uint64_t perft_3_gpu(const std::vector<CPosition>& pos)
{
	const std::size_t size = pos.size();

	thread_local static CPosition* d_pos = nullptr;
	thread_local static uint32_t*  d_res = nullptr;
	if (d_pos == nullptr)
	{
		hipSetDevice(omp_get_thread_num() % 2);
		hipMalloc(&d_pos, sizeof(CPosition) * 100'000'000);
		hipMalloc(&d_res, sizeof(uint32_t) * 100'000'000);
	}
	
	hipMemcpy(d_pos, pos.data(), sizeof(CPosition) * size, hipMemcpyHostToDevice);

	kernel<<<128, 128>>>(d_pos, d_res, size);

	std::vector<uint32_t> result(size);
	hipMemcpy(result.data(), d_res, sizeof(uint32_t) * size, hipMemcpyDeviceToHost);
	//hipFree(d_pos);
	//hipFree(d_res);
	
	return std::accumulate(result.begin(), result.begin() + size, 0ui64);
}


//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
